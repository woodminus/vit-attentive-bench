#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include <vector>
#include "value_aggregation.h"
#include "THC/THCAtomics.cuh"
#include <stdio.h>
#include "utils.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

__global__ void ValueAggregationForwardFunc(float* score, float* value, long* index, float* output, int B, int N, int K, int H, int M, int D) {
