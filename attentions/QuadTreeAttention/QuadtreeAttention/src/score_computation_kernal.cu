#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include <vector>
#include "score_computation.h"
#include <stdio.h>

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32
#define MAX_H 8

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t


template <typename scalar_t>
__global__ void ScoreData(
  torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> query, // B, N1, 4, H, dim
  torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> key, //B, N2, H, d