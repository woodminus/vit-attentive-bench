#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include <vector>
#include "score_computation.h"
#include <stdio.h>

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32
#define MAX_H 8

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t


template <typename scalar_t>
__global__ void ScoreData(
  torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> query, // B, N1, 4, H, dim
  torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> key, //B, N2, H, dim
  torch::PackedTensorAccessor32<long,4,torch::RestrictPtrTraits> index, //B, N1, K*4, H
  torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> output //B, N1, 4, K*4, H
  ){
  extern __shared__ char patch_data_char[];
  
  scalar_t *feat1_data = (scalar_t *)patch_data_char;


  int b = blockIdx.x;
  int n1 = blockIdx.y;
  int f = blockIdx.z;
  
  int ch_off = threadIdx.x;
  
  int D=query.size(4);
  int HD=query.size(3)*D;
  int K=index.size(2);
  for(int ch = ch_off; ch < HD; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
    feat1_data[ch] = query[b][n1][f][ch/D][ch%D];
  }
  __syncthreads();
  
  __shared__ scalar_t score[THREADS_PER_WARP*MAX_H];
  for(int k = ch_off; k < K; k += (WARPS_PER_BLOCK*THREADS_